// #include "../../github/CUDA-by-Example-source-code-for-the-book-s-examples-/common/book.h"

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(void) {
    hipDeviceProp_t prop;

    int count;
    hipGetDeviceCount( &count);
    printf("Device count: %d\n", count);
    for (int i=0; i<count; i++) {
        hipGetDeviceProperties(&prop, i);
        printf("Name: %s\n", prop.name);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);

        printf("##### MEMORY INFO FOR DEVICE %d ---\n", i);
        printf("Total global memory: %ld\n", prop.totalGlobalMem);
        printf("Total constant memory: %ld\n", prop.totalConstMem);
        printf("Max shared memory in byte a single block may use: %ld\n", prop.sharedMemPerBlock);
        printf("Total 32-bits registers per block: %d\n", prop.regsPerBlock);
        printf("Number of threads in a warp: %d\n", prop.warpSize);
        printf("Max pitch allowed for memory copies in bytes: %ld\n", prop.memPitch);     

        printf("##### MP INFORMATION FOR DEVICE %d ---\n", i);
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
        printf("Max threads dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
        printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
        

    }
}