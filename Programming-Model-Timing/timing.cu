#include "hip/hip_runtime.h"
# include <stdio.h>
# include <iostream> 
# include "../common/book.h"
# include <hip/hip_runtime.h>
#define N   4096

using namespace std;

__global__ void add( int *a, int *b, int *c ) {
    int tid = blockIdx.x;    // this thread handles the data at its thread id
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}





int main( void ) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    
    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( &dev_a, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( &dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( &dev_c, N * sizeof(int) ) );

    // fill the arrays 'a' and 'b' on the CPUcreate a function that runs as
    for (int i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
                              hipMemcpyHostToDevice ) );

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Record the start event
    hipEventRecord(start);

    add<<<N,1>>>( dev_a, dev_b, dev_c );

    // Record the stop event
    hipEventRecord(stop);
    // Synchronize to make sure the events have completed
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << "Time taken by the kernel: " << milliseconds << " ms" << std::endl;

    


    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int),
                              hipMemcpyDeviceToHost ) );

    // display the results
    // for (int i=0; i<N; i++) {
    //     printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    // }

    // free the memory allocated on the GPU
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
